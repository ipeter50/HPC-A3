#include "hip/hip_runtime.h"
//extern "C" {

#include <stdio.h>
#include <stdlib.h>
#include "poisson.h"


int
main(int argc, char *argv[]) {
		
	double *h_Uk, *h_Uk1, *h_F;
	double *d_Uk, *d_Uk1, *d_F;
	double *d_Uktop, *d_Uk1top, *d_Ftop;
	double *d_Ukbot, *d_Uk1bot, *d_Fbot;
	double threshold = 0.000000001;
	int N = 200;
	int num_gpu = 1;	
	int max_iter = 10000;
	
	int JACOBI = 1;
	if ( argc >=2 ) N = atoi(argv[1]);
	if ( argc >=3 ) JACOBI = atoi(argv[2]);
	size_t size = (N + 2) * (N + 2) * sizeof(double);	
	printf("Matrix size = %d , Memory = %d\n", N, size);
	
	hipHostMalloc((void **)&h_Uk, size);
	hipHostMalloc((void **)&h_Uk1, size);
	hipHostMalloc((void **)&h_F, size);
	
	init_matrices(h_Uk, h_Uk1, h_F, N);
	//display_mat(h_Uk, N);
	
		
	hipMalloc((void **)&d_Uk, size);	
	hipMalloc((void **)&d_Uk1, size);
	hipMalloc((void **)&d_F, size);
	hipMemcpy(d_Uk, h_Uk, size, hipMemcpyHostToDevice);
	hipMemcpy(d_Uk1, h_Uk1, size, hipMemcpyHostToDevice);
	hipMemcpy(d_F, h_F, size, hipMemcpyHostToDevice);
	

	int k = 0;
	double d = 1000;
	double delta_squared = 1.0/(N+2) * 1.0/(N+2);
	double h = 1.0/4;
	int dimGridX, dimGridY;
	dimGridX = (int)ceil(1.0*N/16);
	dimGridY = dimGridX;
	
	
	while(k<max_iter){
		update_jacobi_gpu<<<1,1>>>(d_Uk, d_Uk1, d_F, N, delta_squared, h);
		//update_jacobi_gpu2<<<dim3(dimGridX, dimGridY),dim3(16,16)>>>(d_Uk, d_Uk1, d_F, N, 					delta_squared, h);
		hipDeviceSynchronize();
		double *tmp = d_Uk;
		d_Uk = d_Uk1;
		d_Uk1 = tmp;
		k = k+1;
	}
	
	
	printf("jacobi_gpu%d\n", JACOBI);
	printf("k = %d\n", k);
	
	
	hipMemcpy(h_Uk, d_Uk, size, hipMemcpyDeviceToHost);
	display_mat(h_Uk, N);
    
		
	hipHostFree(h_Uk);
	hipHostFree(h_Uk1);
	hipHostFree(h_F);
	
	hipFree(d_Uk);
	hipFree(d_Uk1);
	hipFree(d_F);
	
	//display_mat(h_F, N);
	
	
	
	return 0;
}




