#include "hip/hip_runtime.h"
//extern "C" {

#include <stdio.h>
#include <stdlib.h>
#include "poisson.h"


int
main(int argc, char *argv[]) {
		
	double *h_Uk, *h_Uk1, *h_F;
	double *d_Uk, *d_Uk1, *d_F;
	double *d_Uktop, *d_Uk1top, *d_Ftop;
	double *d_Ukbot, *d_Uk1bot, *d_Fbot;
	double threshold = 0.000000001;
	int N = 200;
	int num_gpu = 1;	
	int max_iter = 10000;
	
	int JACOBI = 2;
	if ( argc >=2 ) N = atoi(argv[1]);
	if ( argc >=3 ) JACOBI = atoi(argv[2]);
	size_t size = (N + 2) * (N + 2) * sizeof(double);	
	printf("Matrix size = %d , Memory = %d\n", N, size);

	hipHostMalloc((void **)&h_Uk, size);
	hipHostMalloc((void **)&h_Uk1, size);
	hipHostMalloc((void **)&h_F, size);
	
	init_matrices(h_Uk, h_Uk1, h_F, N);
	//display_mat(h_Uk, N);

	int top_size = (N + 2) * (N + 2)/2;//+N+2;
	//printf("Top: %d\n", top_size);
	//printf("%f\n", h_Uk[top_size]);
		
	hipSetDevice(0);
	hipMalloc((void **)&d_Uktop, size/2);	
	hipMalloc((void **)&d_Uk1top, size/2);
	hipMalloc((void **)&d_Ftop, size/2);		

	hipMemcpy(d_Uktop, h_Uk, size/2, hipMemcpyHostToDevice);
	hipMemcpy(d_Uk1top, h_Uk1, size/2, hipMemcpyHostToDevice);
	hipMemcpy(d_Ftop, h_F, size/2, hipMemcpyHostToDevice);

	
	hipSetDevice(1);

	hipMalloc((void **)&d_Ukbot, size/2);	
	hipMalloc((void **)&d_Uk1bot, size/2);
	hipMalloc((void **)&d_Fbot, size/2);

	hipMemcpy(d_Ukbot, &h_Uk[top_size], size/2, hipMemcpyHostToDevice);
	hipMemcpy(d_Uk1bot, &h_Uk1[top_size], size/2, hipMemcpyHostToDevice);
	hipMemcpy(d_Fbot, &h_F[top_size], size/2, hipMemcpyHostToDevice);
	

	int k = 0;
	double d = 1000;
	double delta_squared = 1.0/(N+2) * 1.0/(N+2);
	double h = 1.0/4;

	int dimGridX, dimGridY;
	dimGridX = (int)ceil(1.0*N/16);
	dimGridY = (int)ceil(1.0*N/32);
	
	while(k<max_iter){
		//update_jacobi_gpu<<<1,1>>>(d_Uk, d_Uk1, d_F, N, delta_squared, h);
		//update_jacobi_gpu2<<<dim3(dimGridX, dimGridY),dim3(16,16)>>>(d_Uk, d_Uk1, d_F, N, delta_squared, h);
		hipSetDevice(0);
		hipDeviceEnablePeerAccess( 1, 0 );
		update_jacobi_2gpu0<<<dim3(dimGridX, dimGridY),dim3(16,16)>>>(d_Uktop, d_Ukbot, d_Uk1top, d_Ftop, N, delta_squared, h);
		hipDeviceSynchronize();
		double *tmptop = d_Uktop;
		d_Uktop = d_Uk1top;
		d_Uk1top = tmptop;

		hipSetDevice(1);
		hipDeviceEnablePeerAccess( 0, 0 );
		update_jacobi_2gpu1<<<dim3(dimGridX, dimGridY),dim3(16,16)>>>(d_Ukbot, d_Uktop, d_Uk1bot, d_Fbot, N, delta_squared, h);
		hipDeviceSynchronize();
		double *tmpbot = d_Ukbot;
		d_Ukbot = d_Uk1bot;
		d_Uk1bot = tmpbot;
		
		
		
		k = k+1;
	}
	
	
	printf("jacobi_gpu%d\n", JACOBI);
	printf("k = %d\n", k);
	
	hipDeviceDisablePeerAccess(0);
	hipDeviceDisablePeerAccess(1);

	hipSetDevice(0);
	hipMemcpy(h_Uk, d_Uktop, size/2, hipMemcpyDeviceToHost);
	hipSetDevice(1);
	hipMemcpy(&h_Uk[top_size], d_Ukbot, size/2, hipMemcpyDeviceToHost);
	//display_mat(h_Uk, N);
    
		
	hipHostFree(h_Uk);
	hipHostFree(h_Uk1);
	hipHostFree(h_F);
	
	hipFree(d_Uk);
	hipFree(d_Uk1);
	hipFree(d_F);
	
	//display_mat(h_F, N);
	
	
	
	return 0;
}




