#include "hip/hip_runtime.h"
extern "C"{

#include <stdio.h>
#include "lib_kernels.h"


__global__ void
kernel_gpu1(double *d_A, double *d_B, double *d_C, int m, int n, int k){

	int i, j, l;
	for(i = 0; i < m; i++){
		for(j = 0; j < n; j++){
			d_C[i * m + j] = 0;
	    	}
	    }    


	for(j = 0; j < n; j++){
		for(i = 0; i < m; i++){
			double tmp = 0.0;
			for(l = 0; l < k; l++){
				tmp += d_A[i*m +l] * d_B[l*k+j];
			}
			d_C[i*m +j] = tmp;
		}
	}

}


__global__ void
kernel_gpu2(double *d_A, double *d_B, double *d_C, int m, int n, int k){
	int i,j;
	j = threadIdx.x + blockIdx.x * blockDim.x;
	i = threadIdx.y + blockIdx.y * blockDim.y;
	if(j<n && i<m){
		double tmp = 0.0;
		for(int l = 0; l < k; l++){
			tmp += d_A[i*k+l] * d_B[j+n*l];
		}
		d_C[i*n + j] = tmp;
	}
}


__global__ void
kernel_gpu3(double *d_A, double *d_B, double *d_C, int m, int n, int k){
	int i,j;
	j = 1 * (threadIdx.x + blockIdx.x * blockDim.x);
	i = 2 * (threadIdx.y + blockIdx.y * blockDim.y);
	if(j<n && i<m){
		double tmp = 0.0;
		double tmp2 = 0.0;
		for(int l = 0; l < k; l++){
			tmp += d_A[i*k+l] * d_B[j+n*l];
			tmp2 += d_A[(i+1)*k + l] * d_B[j+n*l];
		}
		d_C[i*n + j] = tmp;
		d_C[(i+1)*n + j] = tmp2;
	}
}


}
